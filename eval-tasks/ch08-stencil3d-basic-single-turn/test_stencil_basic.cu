// test_stencil_basic.cu
#include <hip/hip_runtime.h>
#include <cassert>
#include <cstdio>
#include <vector>
#include <random>
#include <algorithm>
#include <cmath>

extern __global__ void stencil3d_basic_student(const float*, float*, int, float,float,float,float,float,float,float);

static void cpu_oracle(
    const std::vector<float>& in,
    std::vector<float>& out,
    int N,
    float c0,float c1,float c2,float c3,float c4,float c5,float c6)
{
    auto idx = [N](int I,int J,int K){ return (I*N + J)*N + K; };
    if (N <= 0) return;
    for (int i=0;i<N;i++){
        for (int j=0;j<N;j++){
            for (int k=0;k<N;k++){
                bool interior = (i>0&&i<N-1)&&(j>0&&j<N-1)&&(k>0&&k<N-1);
                if(!interior){
                    out[idx(i,j,k)] = in[idx(i,j,k)];
                }else{
                    float ctr=in[idx(i,j,k)];
                    float xm=in[idx(i,j,k-1)], xp=in[idx(i,j,k+1)];
                    float ym=in[idx(i,j-1,k)], yp=in[idx(i,j+1,k)];
                    float zm=in[idx(i-1,j,k)], zp=in[idx(i+1,j,k)];
                    out[idx(i,j,k)] = c0*ctr + c1*xm + c2*xp + c3*ym + c4*yp + c5*zm + c6*zp;
                }
            }
        }
    }
}

static void run_case(int N, dim3 block)
{
    const float c0=0.5f, c1=0.1f, c2=0.1f, c3=0.05f, c4=0.05f, c5=0.1f, c6=0.1f;

    size_t count = (N<=0)?0: (size_t)N*N*N;
    std::vector<float> h_in(count), h_out_gpu(count, 1337.0f), h_out_cpu(count, 0.0f);
    // Adversarial fill
    for (size_t t=0;t<count;t++){
        // deterministic pattern; catches off-by-one, striding, etc.
        h_in[t] = float((t*1315423911u) ^ 0x9e3779b9u) * 1.0e-9f;
    }

    float *d_in=nullptr, *d_out=nullptr;
    hipMalloc(&d_in, std::max<size_t>(count,1)*sizeof(float));
    hipMalloc(&d_out,std::max<size_t>(count,1)*sizeof(float));
    hipMemcpy(d_in, h_in.data(), count*sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_out, 0xCD, count*sizeof(float)); // canary

    dim3 grid((N+block.x-1)/block.x,
              (N+block.y-1)/block.y,
              (N+block.z-1)/block.z);

    // CPU oracle
    cpu_oracle(h_in, h_out_cpu, N, c0,c1,c2,c3,c4,c5,c6);

    // GPU implementation (student or reference)
    stencil3d_basic_student<<<grid,block>>>(d_in, d_out, N, c0,c1,c2,c3,c4,c5,c6);
    hipDeviceSynchronize();
    hipMemcpy(h_out_gpu.data(), d_out, count*sizeof(float), hipMemcpyDeviceToHost);
    // Compare with relative tolerance
    for (size_t i=0;i<count;i++){
        float rel_err = std::fabs(h_out_gpu[i]-h_out_cpu[i]) / (std::fabs(h_out_cpu[i]) + 1e-6f);
        if (rel_err > 1e-5f){
            fprintf(stderr,"MISMATCH at %zu: got %f, exp %f\n", i, h_out_gpu[i], h_out_cpu[i]);
            exit(1);
        }
    }

    // Input immutability
    std::vector<float> h_in_after(count, 0.0f);
    hipMemcpy(h_in_after.data(), d_in, count*sizeof(float), hipMemcpyDeviceToHost);
    if (h_in_after != h_in){
        fprintf(stderr,"Input array was modified!\n");
        exit(1);
    }

    hipFree(d_in); hipFree(d_out);
    printf("  N=%d ... OK\n", N);
}

int main(){
    printf("stencil3d-basic-single-turn tests\n");

    // Multiple block sizes (odd + common)
    const dim3 blocks[] = { dim3(8,8,8), dim3(4,8,16) };
    for (auto b : blocks){
        printf("Block = (%d,%d,%d)\n", b.x, b.y, b.z);
        // Edge and adversarial sizes
        int Ns[] = {0,1,2,3,4,7,8,16,17,32};
        for (int N: Ns){
            run_case(N, b);
        }
    }
    printf("All tests passed.\n");
    return 0;
}