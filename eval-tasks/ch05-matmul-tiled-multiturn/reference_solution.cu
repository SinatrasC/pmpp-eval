// reference_solution.cu
// Correct reference implementation for shared-memory tiled matmul.

#include <hip/hip_runtime.h>

extern "C" void launch_reference(const float* A, const float* B, float* C,
                                 int M, int N, int K, int blockSize);

__global__ void matmul_tiled_ref(const float* __restrict__ A,
                                 const float* __restrict__ B,
                                 float* __restrict__ C,
                                 int M, int N, int K)
{
    const int TILE = 16;

    int row = blockIdx.y * TILE + threadIdx.y;
    int col = blockIdx.x * TILE + threadIdx.x;

    __shared__ float As[TILE][TILE];
    __shared__ float Bs[TILE][TILE];

    float acc = 0.0f;
    int tiles = (N + TILE - 1) / TILE;

    for (int t = 0; t < tiles; ++t) {
        int aRow = row;
        int aCol = t * TILE + threadIdx.x;
        int bRow = t * TILE + threadIdx.y;
        int bCol = col;

        As[threadIdx.y][threadIdx.x] =
            (aRow < M && aCol < N) ? A[aRow * N + aCol] : 0.0f;

        Bs[threadIdx.y][threadIdx.x] =
            (bRow < N && bCol < K) ? B[bRow * K + bCol] : 0.0f;

        __syncthreads();

        #pragma unroll
        for (int k = 0; k < TILE; ++k) {
            acc += As[threadIdx.y][k] * Bs[k][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < M && col < K) {
        C[row * K + col] = acc;
    }
}

extern "C" void launch_reference(const float* A, const float* B, float* C,
                                 int M, int N, int K, int /*blockSize*/)
{
    dim3 block(16, 16);
    dim3 grid((K + 15) / 16, (M + 15) / 16);
    matmul_tiled_ref<<<grid, block>>>(A, B, C, M, N, K);
}