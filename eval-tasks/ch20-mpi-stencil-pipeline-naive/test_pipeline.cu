#include <hip/hip_runtime.h>
#include <cstdio>
#include <vector>
#include <algorithm>
#include <cmath>
#include <cassert>

extern "C" void mpi_stencil_pipeline_naive(const float* d_in_full,
                                           float* d_out_full,
                                           int dimx,int dimy,int dimz_total,
                                           int procs);

static inline size_t idx3(int i,int j,int k,int dx,int dy){ return (size_t(k)*dy + j)*dx + i; }
static void ck(hipError_t e,const char* m){ if(e!=hipSuccess){fprintf(stderr,"CUDA %s: %s\n",m,hipGetErrorString(e)); std::exit(2);} }

static void fill_pattern(std::vector<float>& a){
    for(size_t t=0;t<a.size();++t){
        a[t] = 0.3f*sinf(0.01f*t) + 0.7f*cosf(0.013f*(t%271)) + 0.001f*float((t*17)%101);
    }
}

static void cpu_oracle(const std::vector<float>& in, std::vector<float>& out,
                       int dimx,int dimy,int dimz)
{
    const int R=4;
    auto inside=[&](int i,int j,int k){ return (i>=0&&i<dimx&&j>=0&&j<dimy&&k>=0&&k<dimz); };

    out.resize(in.size());
    out = in; // copy-through by default

    const float w0=0.5f, w1=0.10f, w2=0.05f, w3=0.025f, w4=0.0125f;
    const float w[5]={w0,w1,w2,w3,w4};

    for(int k=0;k<dimz;++k){
        for(int j=0;j<dimy;++j){
            for(int i=0;i<dimx;++i){
                // copy-through on any global face within radius
                if(i<R || i>=dimx-R || j<R || j>=dimy-R || k<R || k>=dimz-R){
                    out[idx3(i,j,k,dimx,dimy)] = in[idx3(i,j,k,dimx,dimy)];
                    continue;
                }
                float acc = w[0]*in[idx3(i,j,k,dimx,dimy)];
                for(int d=1; d<=4; ++d){
                    acc += w[d]*( in[idx3(i-d,j,k,dimx,dimy)] + in[idx3(i+d,j,k,dimx,dimy)]
                                + in[idx3(i,j-d,k,dimx,dimy)] + in[idx3(i,j+d,k,dimx,dimy)]
                                + in[idx3(i,j,k-d,dimx,dimy)] + in[idx3(i,j,k+d,dimx,dimy)] );
                }
                out[idx3(i,j,k,dimx,dimy)] = acc;
            }
        }
    }
}

static bool almost_equal(const std::vector<float>& a, const std::vector<float>& b){
    if(a.size()!=b.size()) return false;
    for(size_t i=0;i<a.size();++i){
        float A=a[i], B=b[i];
        float diff=fabsf(A-B);
        float tol=1e-5f + 1e-5f*std::max(fabsf(A),fabsf(B));
        if(diff>tol) return false;
    }
    return true;
}

int main(){
    printf("ch20-mpi-stencil-pipeline-naive tests\n");

    struct C{int x,y,z,procs;};
    const C cases[] = {
        {16,16,16,2},
        {24,20,24,3},
        {32,24,32,2},
        {48,48,40,2}
    };

    const size_t GUARD=4096;
    const float SENT=12345.0f;

    int total=0, pass=0;

    for(const auto& cs : cases){
        ++total;
        int dimx=cs.x, dimy=cs.y, dimz=cs.z, procs=cs.procs;
        assert(dimz%procs==0);

        size_t N = size_t(dimx)*dimy*dimz;

        std::vector<float> hin(N), href(N), hout(N);
        fill_pattern(hin);
        cpu_oracle(hin, href, dimx,dimy,dimz);

        std::vector<float> h_in_guard (N+2*GUARD, SENT);
        std::vector<float> h_out_guard(N+2*GUARD, SENT);
        std::copy(hin.begin(), hin.end(), h_in_guard.begin()+GUARD);

        float *d_in_all=nullptr, *d_out_all=nullptr;
        ck(hipMalloc(&d_in_all,  (N+2*GUARD)*sizeof(float)),"malloc in");
        ck(hipMalloc(&d_out_all, (N+2*GUARD)*sizeof(float)),"malloc out");
        ck(hipMemcpy(d_in_all,  h_in_guard.data(),  (N+2*GUARD)*sizeof(float), hipMemcpyHostToDevice),"H2D in");
        ck(hipMemcpy(d_out_all, h_out_guard.data(), (N+2*GUARD)*sizeof(float), hipMemcpyHostToDevice),"H2D out");

        float* d_in  = d_in_all  + GUARD;
        float* d_out = d_out_all + GUARD;

        mpi_stencil_pipeline_naive(d_in, d_out, dimx,dimy,dimz, procs);

        ck(hipMemcpy(h_out_guard.data(), d_out_all, (N+2*GUARD)*sizeof(float), hipMemcpyDeviceToHost),"D2H out");
        std::copy(h_out_guard.begin()+GUARD, h_out_guard.begin()+GUARD+N, hout.begin());

        auto guard_ok=[&](const std::vector<float>& g){
            for(size_t t=0;t<GUARD;t++){
                if(g[t]!=SENT || g[g.size()-1-t]!=SENT) return false;
            } return true;
        };

        bool ok = almost_equal(hout, href) && guard_ok(h_out_guard);
        printf("Case %3dx%3dx%3d P=%d -> %s\n", dimx,dimy,dimz, procs, ok?"OK":"FAIL");
        if(ok) ++pass;

        hipFree(d_in_all); hipFree(d_out_all);
    }

    printf("Summary: %d/%d passed\n", pass,total);
    return (pass==total)?0:1;
}