#include <hip/hip_runtime.h>
#include <cstdio>
#include <vector>
#include <algorithm>
#include <cmath>
#include <cassert>

static inline __host__ __device__
size_t idx3(int i,int j,int k,int dx,int dy){ return (size_t(k)*dy + j)*dx + i; }

// ====================== Stencil Kernels (25-pt, R=4) ======================
__global__ void k_stage1_boundary(const float* __restrict__ in,
                                  float* __restrict__ out,
                                  int dimx,int dimy,int dz_local,
                                  int z_global_beg, int dimz_total)
{
    const int R=4;
    const int zOwnedBeg = 4;
    const int zOwnedEnd = 4 + dz_local - 1;

    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z; // local z including halos
    if(i>=dimx || j>=dimy || k>=dz_local+8) return;

    // boundary planes: first 4 and last 4 owned planes
    bool isBoundaryZ = (k>=zOwnedBeg && k<=zOwnedBeg+3) ||
                       (k>=zOwnedEnd-3 && k<=zOwnedEnd);
    if(!isBoundaryZ) return;

    // copy-through on x/y faces always
    if(i< R || i>=dimx-R || j< R || j>=dimy-R){
        out[idx3(i,j,k,dimx,dimy)] = in[idx3(i,j,k,dimx,dimy)];
        return;
    }

    // global z index for this local k
    int z_global = z_global_beg + (k - 4);

    // if the 4-neighborhood in z would leave the global domain => copy-through
    if(z_global - 4 < 0 || z_global + 4 >= dimz_total){
        out[idx3(i,j,k,dimx,dimy)] = in[idx3(i,j,k,dimx,dimy)];
        return;
    }

    const float w0=0.5f, w1=0.10f, w2=0.05f, w3=0.025f, w4=0.0125f;
    const float w[5]={w0,w1,w2,w3,w4};

    size_t p = idx3(i,j,k,dimx,dimy);
    float acc = w[0]*in[p];
    #pragma unroll
    for(int d=1; d<=4; ++d){
        acc += w[d]*( in[idx3(i-d,j,k,dimx,dimy)] + in[idx3(i+d,j,k,dimx,dimy)]
                    + in[idx3(i,j-d,k,dimx,dimy)] + in[idx3(i,j+d,k,dimx,dimy)]
                    + in[idx3(i,j,k-d,dimx,dimy)] + in[idx3(i,j,k+d,dimx,dimy)] );
    }
    out[p] = acc;
}

__global__ void k_stage2_interior(const float* __restrict__ in,
                                  float* __restrict__ out,
                                  int dimx,int dimy,int dz_local)
{
    const int R=4;
    const int zOwnedBeg = 4;
    const int zOwnedEnd = 4 + dz_local - 1;

    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z;
    if(i>=dimx || j>=dimy || k>=dz_local+8) return;

    // interior only (skip 4 planes near each local z end)
    if(!(k >= zOwnedBeg+4 && k <= zOwnedEnd-4)) return;

    if(i< R || i>=dimx-R || j< R || j>=dimy-R){
        out[idx3(i,j,k,dimx,dimy)] = in[idx3(i,j,k,dimx,dimy)];
        return;
    }

    const float w0=0.5f, w1=0.10f, w2=0.05f, w3=0.025f, w4=0.0125f;
    const float w[5]={w0,w1,w2,w3,w4};

    size_t p = idx3(i,j,k,dimx,dimy);
    float acc = w[0]*in[p];
    #pragma unroll
    for(int d=1; d<=4; ++d){
        acc += w[d]*( in[idx3(i-d,j,k,dimx,dimy)] + in[idx3(i+d,j,k,dimx,dimy)]
                    + in[idx3(i,j-d,k,dimx,dimy)] + in[idx3(i,j+d,k,dimx,dimy)]
                    + in[idx3(i,j,k-d,dimx,dimy)] + in[idx3(i,j,k+d,dimx,dimy)] );
    }
    out[p] = acc;
}

static void launch_stage1(const float* d_in, float* d_out,
                          int dimx,int dimy,int dz_local,
                          int z_global_beg, int dimz_total)
{
    dim3 block(8,8,8);
    dim3 grid((dimx+7)/8, (dimy+7)/8, ((dz_local+8)+7)/8);
    k_stage1_boundary<<<grid,block>>>(d_in, d_out, dimx,dimy,dz_local, z_global_beg, dimz_total);
}

static void launch_stage2(const float* d_in, float* d_out,
                          int dimx,int dimy,int dz_local)
{
    dim3 block(8,8,8);
    dim3 grid((dimx+7)/8, (dimy+7)/8, ((dz_local+8)+7)/8);
    k_stage2_interior<<<grid,block>>>(d_in, d_out, dimx,dimy,dz_local);
}

// ====================== Pack / Unpack (4 planes per side) ==================
static inline __host__ __device__
size_t pack_idx(int p,int j,int i,int dx,int dy){ return (size_t(p)*dy + j)*dx + i; }

__global__ void k_pack(const float* __restrict__ slab_out,
                       int dimx,int dimy,int dz_local,
                       float* __restrict__ left_send,
                       float* __restrict__ right_send)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    if(i>=dimx || j>=dimy) return;

    const int zOwnedBeg = 4;
    const int zOwnedEnd = 4 + dz_local - 1;

    #pragma unroll
    for(int p=0;p<4;++p){
        int kL = zOwnedBeg + p;
        int kR = (zOwnedEnd - 3) + p;
        size_t sL = idx3(i,j,kL,dimx,dimy);
        size_t sR = idx3(i,j,kR,dimx,dimy);
        size_t d  = pack_idx(p,j,i,dimx,dimy);
        left_send [d] = slab_out[sL];
        right_send[d] = slab_out[sR];
    }
}

__global__ void k_unpack(float* __restrict__ slab_out,
                         int dimx,int dimy,int dz_local,
                         const float* __restrict__ left_recv,
                         const float* __restrict__ right_recv)
{
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    if(i>=dimx || j>=dimy) return;

    const int kHaloL = 0;
    const int kHaloR = dz_local + 4;

    #pragma unroll
    for(int p=0;p<4;++p){
        size_t s = pack_idx(p,j,i,dimx,dimy);
        if (left_recv)  slab_out[idx3(i,j,kHaloL+p,dimx,dimy)] = left_recv [s];
        if (right_recv) slab_out[idx3(i,j,kHaloR+p,dimx,dimy)] = right_recv[s];
    }
}

void stencil25_stage1_boundary(const float* d_in, float* d_out,
                               int dimx,int dimy,int dz_local,
                               int z_global_beg, int dimz_total)
{
    launch_stage1(d_in, d_out, dimx,dimy,dz_local, z_global_beg, dimz_total);
    hipDeviceSynchronize();
}
void stencil25_stage2_interior(const float* d_in, float* d_out,
                               int dimx,int dimy,int dz_local)
{
    launch_stage2(d_in, d_out, dimx,dimy,dz_local);
    hipDeviceSynchronize();
}
void halo_pack_boundaries(const float* d_slab_out,
                          int dimx,int dimy,int dz_local,
                          float* d_left_send, float* d_right_send)
{
    dim3 b(16,16); dim3 g((dimx+15)/16,(dimy+15)/16);
    k_pack<<<g,b>>>(d_slab_out, dimx,dimy,dz_local, d_left_send,d_right_send);
    hipDeviceSynchronize();
}
void halo_unpack_to_halos(float* d_slab_out,
                          int dimx,int dimy,int dz_local,
                          const float* d_left_recv, const float* d_right_recv)
{
    dim3 b(16,16); dim3 g((dimx+15)/16,(dimy+15)/16);
    k_unpack<<<g,b>>>(d_slab_out, dimx,dimy,dz_local, d_left_recv,d_right_recv);
    hipDeviceSynchronize();
}

// ====================== Pipeline Orchestration (reference) =================
static void ck(hipError_t e,const char* m){ if(e!=hipSuccess){fprintf(stderr,"CUDA %s: %s\n",m,hipGetErrorString(e)); std::exit(2);} }

__global__ void k_scatter_from_full(const float* __restrict__ d_in_full,
                                    float* __restrict__ d_slab_in,
                                    int dimx,int dimy,int z0,int dz)
{
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    int j=blockIdx.y*blockDim.y+threadIdx.y;
    int t=blockIdx.z*blockDim.z+threadIdx.z; // local owned z [0..dz-1]
    if(i>=dimx||j>=dimy||t>=dz) return;
    int k_local = 4 + t;
    int k_full  = z0 + t;
    d_slab_in[idx3(i,j,k_local,dimx,dimy)] =
        d_in_full[idx3(i,j,k_full,dimx,dimy)];
}

__global__ void k_gather_to_full(const float* __restrict__ d_slab_out,
                                 float* __restrict__ d_out_full,
                                 int dimx,int dimy,int z0,int dz)
{
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    int j=blockIdx.y*blockDim.y+threadIdx.y;
    int t=blockIdx.z*blockDim.z+threadIdx.z; // local owned z [0..dz-1]
    if(i>=dimx||j>=dimy||t>=dz) return;
    int k_local = 4 + t;
    int k_full  = z0 + t;
    d_out_full[idx3(i,j,k_full,dimx,dimy)] =
        d_slab_out[idx3(i,j,k_local,dimx,dimy)];
}

extern "C" void mpi_stencil_pipeline_naive(const float* d_in_full,
                                           float* d_out_full,
                                           int dimx,int dimy,int dimz_total,
                                           int procs)
{
    assert(procs>=1 && dimz_total%procs==0);
    int dz = dimz_total / procs;
    size_t Nplane = size_t(dimx)*dimy;
    size_t Nslab  = Nplane*(dz+8);
    size_t Npack  = Nplane*4;

    // Per-slab buffers
    std::vector<float*> d_in (procs,nullptr), d_out(procs,nullptr);
    std::vector<float*> d_Ls (procs,nullptr), d_Rs(procs,nullptr);
    std::vector<float*> d_Lr (procs,nullptr), d_Rr(procs,nullptr);

    for(int r=0;r<procs;++r){
        ck(hipMalloc(&d_in[r],  Nslab*sizeof(float)),"malloc slab in");
        ck(hipMalloc(&d_out[r], Nslab*sizeof(float)),"malloc slab out");
        ck(hipMalloc(&d_Ls[r],  Npack*sizeof(float)),"malloc Ls");
        ck(hipMalloc(&d_Rs[r],  Npack*sizeof(float)),"malloc Rs");
        ck(hipMalloc(&d_Lr[r],  Npack*sizeof(float)),"malloc Lr");
        ck(hipMalloc(&d_Rr[r],  Npack*sizeof(float)),"malloc Rr");
        // Seed both in/out with input copy for pass-through faces
        ck(hipMemset(d_in[r],  0, Nslab*sizeof(float)),"memset in");
        ck(hipMemset(d_out[r], 0, Nslab*sizeof(float)),"memset out");
    }

    // Scatter global input into local slabs (owned planes at k_local=[4..])
    {
        dim3 b(8,8,8);
        for(int r=0;r<procs;++r){
            int z0 = r*dz;
            dim3 g((dimx+7)/8, (dimy+7)/8, (dz+7)/8);
            k_scatter_from_full<<<g,b>>>(d_in_full, d_in[r], dimx,dimy, z0,dz);
        }
        ck(hipDeviceSynchronize(),"sync scatter");
        // Copy in -> out initially (so untouched positions remain pass-through)
        for(int r=0;r<procs;++r){
            ck(hipMemcpy(d_out[r], d_in[r], Nslab*sizeof(float), hipMemcpyDeviceToDevice), "seed out");
        }
    }

    // --- Fill halos in IN FIRST (pack/exchange/unpack) ---
    // Pack boundary planes FROM IN (owned edge planes k_local=4..7 and k_local=dz-3..dz)
    for(int r=0;r<procs;++r){
        halo_pack_boundaries(d_in[r], dimx,dimy,dz, d_Ls[r], d_Rs[r]);
    }

    // Simulate MPI exchange (device->device copies)
    for(int r=0;r<procs;++r){
        int left  = (r>0)        ? r-1 : -1;
        int right = (r<procs-1)  ? r+1 : -1;
        if(left  >=0){ ck(hipMemcpy(d_Rr[left], d_Ls[r], Npack*sizeof(float), hipMemcpyDeviceToDevice),"xchg L->left.Rr"); }
        if(right >=0){ ck(hipMemcpy(d_Lr[right],d_Rs[r], Npack*sizeof(float), hipMemcpyDeviceToDevice),"xchg R->right.Lr"); }
    }

    // Unpack RECV buffers into IN halos; only sides that exist
    for(int r=0;r<procs;++r){
        const float* L = (r>0)        ? d_Lr[r] : nullptr;
        const float* R = (r<procs-1)  ? d_Rr[r] : nullptr;
        halo_unpack_to_halos(d_in[r], dimx,dimy,dz, L, R);
    }

    // --- Now Stage-1: boundary update on OUT, reading halos from IN ---
    for(int r=0;r<procs;++r){
        int z0 = r*dz;
        stencil25_stage1_boundary(d_in[r], d_out[r], dimx,dimy,dz, z0, dimz_total);
    }

    // Stage-2 interior update on OUT (using IN as input)
    for(int r=0;r<procs;++r){
        stencil25_stage2_interior(d_in[r], d_out[r], dimx,dimy,dz);
    }

    // Gather owned planes from OUT back to d_out_full
    {
        dim3 b(8,8,8);
        for(int r=0;r<procs;++r){
            int z0=r*dz; dim3 g((dimx+7)/8,(dimy+7)/8,(dz+7)/8);
            k_gather_to_full<<<g,b>>>(d_out[r], d_out_full, dimx,dimy, z0,dz);
        }
        ck(hipDeviceSynchronize(),"sync gather");
    }

    for(int r=0;r<procs;++r){
        hipFree(d_in[r]); hipFree(d_out[r]); hipFree(d_Ls[r]); hipFree(d_Rs[r]); hipFree(d_Lr[r]); hipFree(d_Rr[r]);
    }
}