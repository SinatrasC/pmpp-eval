// ch18-energy-scatter-single / test_energy_scatter.cu
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cmath>
#include <cstdio>
#include <random>
#include <vector>

#ifndef CHUNK_SIZE
#define CHUNK_SIZE 256
#endif

// Kernel under test:
extern "C" __global__
void cenergyScatterKernel(float* energygrid, dim3 grid, float gridspacing, float z,
                          int atoms_in_chunk, int start_atom);

// Forward declare the constant memory symbol
extern __constant__ float atoms[CHUNK_SIZE * 4];

static void ck(hipError_t e, const char* m){
    if(e != hipSuccess){ std::fprintf(stderr, "CUDA %s: %s\n", m, hipGetErrorString(e)); std::exit(2); }
}

// CPU oracle: compute full-slice energy in double for stability, then cast to float
static void cpu_oracle_slice(std::vector<float>& out, dim3 grid, float gridspacing, float z,
                             const std::vector<float>& h_atoms /* len = 4*N */) {
    const int N = (int)h_atoms.size() / 4;
    out.assign((size_t)grid.x * grid.y * grid.z, 0.0f);

    const int k = int(z / gridspacing);
    for (int j = 0; j < (int)grid.y; ++j) {
        double y = gridspacing * (double)j;
        for (int i = 0; i < (int)grid.x; ++i) {
            double x = gridspacing * (double)i;
            double sum = 0.0;
            for (int a = 0; a < N; ++a) {
                double ax = (double)h_atoms[4*a + 0];
                double ay = (double)h_atoms[4*a + 1];
                double az = (double)h_atoms[4*a + 2];
                double q  = (double)h_atoms[4*a + 3];
                double dx = x - ax, dy = y - ay, dz = z - az;
                double denom = std::sqrt(dx*dx + dy*dy + dz*dz);
                sum += q / std::max(denom, 1e-18);
            }
            size_t idx = (size_t)grid.x * grid.y * k + (size_t)grid.x * j + (size_t)i;
            out[idx] = (float)sum;
        }
    }
}

static bool almost_equal(const std::vector<float>& a, const std::vector<float>& b,
                         float abs_eps = 1e-5f, float rel_eps = 1e-5f){
    if (a.size() != b.size()) return false;
    for (size_t i=0;i<a.size();++i){
        float x=a[i], y=b[i];
        float diff = std::fabs(x-y);
        if (!(diff <= abs_eps || diff <= rel_eps * std::max(1.0f, std::max(std::fabs(x), std::fabs(y))))) {
            // debug on first mismatch
            // std::fprintf(stderr, "Mismatch @%zu: got=%.8f ref=%.8f\n", i, x, y);
            return false;
        }
    }
    return true;
}

int main(){
    std::printf("ch18-energy-scatter-single tests\n");

    // Test matrix
    struct GridCfg { dim3 g; float dz; const char* name; };
    const GridCfg grids[] = {
        {{8,8,3}, 1.0f,  "8x8x3"},
        {{31,17,4},1.0f, "31x17x4"},
        {{64,33,4},1.0f, "64x33x4"},
    };
    const int atom_counts[] = {0, 1, 17, CHUNK_SIZE-3, CHUNK_SIZE, CHUNK_SIZE+5};

    const size_t GUARD = 1024;
    const float  SENT  = 1337.0f;
    std::mt19937 rng(42);
    std::uniform_real_distribution<float> pos01(0.0f, 1.0f);
    std::uniform_real_distribution<float> charge(-2.0f, 3.0f);

    int total=0, passed=0;

    for (auto cfg : grids){
        float gridspacing = 1.0f;
        float z = gridspacing * cfg.dz;    // choose the k = dz slice

        for (int NA : atom_counts){
            // Build atoms AoS [x,y,z,q]
            std::vector<float> h_atoms(4 * NA);
            for (int a=0; a<NA; ++a){
                // Put atoms loosely within [0,grid) region, arbitrary z around slice
                h_atoms[4*a+0] = pos01(rng) * (cfg.g.x * gridspacing);
                h_atoms[4*a+1] = pos01(rng) * (cfg.g.y * gridspacing);
                h_atoms[4*a+2] = (pos01(rng) * 2.0f - 0.5f) * (cfg.g.z * gridspacing); // allow near/far
                h_atoms[4*a+3] = charge(rng);
            }

            // CPU oracle
            std::vector<float> h_ref;
            cpu_oracle_slice(h_ref, cfg.g, gridspacing, z, h_atoms);

            // Device buffer with guard canaries
            const size_t Nout = (size_t)cfg.g.x * cfg.g.y * cfg.g.z;
            std::vector<float> h_out_guard(Nout + 2*GUARD, SENT);
            float* d_out_all = nullptr;
            ck(hipMalloc(&d_out_all, (Nout + 2*GUARD) * sizeof(float)), "malloc out");
            ck(hipMemcpy(d_out_all, h_out_guard.data(), (Nout + 2*GUARD)*sizeof(float),
                          hipMemcpyHostToDevice), "H2D canary");
            float* d_out = d_out_all + GUARD;

            auto guard_ok = [&](const std::vector<float>& g){
                for (size_t i=0;i<GUARD;i++){
                    if (g[i] != SENT) return false;
                    if (g[g.size()-1-i] != SENT) return false;
                }
                return true;
            };

            // --- Launch in CHUNKS in natural order ---
            // zero interior
            ck(hipMemset(d_out, 0, Nout * sizeof(float)), "memset out");

            int chunks = (NA + CHUNK_SIZE - 1) / CHUNK_SIZE;
            for (int c=0; c<chunks; ++c){
                int start = c * CHUNK_SIZE;
                int count = std::min(CHUNK_SIZE, NA - start);
                if (count <= 0) break;

                ck(hipMemcpyToSymbol(HIP_SYMBOL(atoms), h_atoms.data() + 4*start,
                                      count * 4 * sizeof(float), 0, hipMemcpyHostToDevice),
                   "H2C atoms");
                dim3 block(256);
                dim3 grid((count + block.x - 1) / block.x);
                cenergyScatterKernel<<<grid,block>>>(d_out, cfg.g, gridspacing, z, count, start);
                ck(hipGetLastError(), "launch");
                ck(hipDeviceSynchronize(), "sync");
            }
            // Download
            ck(hipMemcpy(h_out_guard.data(), d_out_all, (Nout+2*GUARD)*sizeof(float),
                          hipMemcpyDeviceToHost), "D2H");
            std::vector<float> h_out(Nout);
            std::copy(h_out_guard.begin()+GUARD, h_out_guard.begin()+GUARD+Nout, h_out.begin());

            bool ok = almost_equal(h_out, h_ref) && guard_ok(h_out_guard);
            std::printf("Grid %-8s NA=%5d order=seq   -> %s\n", cfg.name, NA, ok?"OK":"FAIL");
            ++total; if (ok) ++passed;

            // --- Re-run with shuffled chunk order to test commutativity / atomicity ---
            ck(hipMemcpy(d_out_all, h_out_guard.data(), (Nout+2*GUARD)*sizeof(float),
                          hipMemcpyHostToDevice), "H2D reset canary");
            ck(hipMemset(d_out, 0, Nout * sizeof(float)), "memset out");

            std::vector<int> ord(chunks);
            for (int i=0;i<chunks;i++) ord[i]=i;
            std::shuffle(ord.begin(), ord.end(), rng);

            for (int idx=0; idx<chunks; ++idx){
                int c = ord[idx];
                int start = c * CHUNK_SIZE;
                int count = std::min(CHUNK_SIZE, NA - start);
                if (count <= 0) continue;

                ck(hipMemcpyToSymbol(HIP_SYMBOL(atoms), h_atoms.data() + 4*start,
                                      count * 4 * sizeof(float), 0, hipMemcpyHostToDevice),
                   "H2C atoms");
                dim3 block(256);
                dim3 grid((count + block.x - 1) / block.x);
                cenergyScatterKernel<<<grid,block>>>(d_out, cfg.g, gridspacing, z, count, start);
                ck(hipGetLastError(), "launch");
                ck(hipDeviceSynchronize(), "sync");
            }
            ck(hipMemcpy(h_out_guard.data(), d_out_all, (Nout+2*GUARD)*sizeof(float),
                          hipMemcpyDeviceToHost), "D2H shuffled");
            std::copy(h_out_guard.begin()+GUARD, h_out_guard.begin()+GUARD+Nout, h_out.begin());

            ok = almost_equal(h_out, h_ref) && guard_ok(h_out_guard);
            std::printf("Grid %-8s NA=%5d order=shuf  -> %s\n", cfg.name, NA, ok?"OK":"FAIL");
            ++total; if (ok) ++passed;

            hipFree(d_out_all);
        }
    }

    std::printf("Summary: %d / %d passed\n", passed, total);
    return (passed==total)?0:1;
}