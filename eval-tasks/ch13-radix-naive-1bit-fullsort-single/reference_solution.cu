// reference_solution.cu
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <cstdio>
#include <vector>
#include <algorithm>

static inline void CK(hipError_t e, const char* m){
    if(e != hipSuccess){
        std::fprintf(stderr, "CUDA %s: %s\n", m, hipGetErrorString(e));
        std::exit(2);
    }
}

constexpr int BLOCK = 256;

// K1: flagsZero[i] = 1 if ((x >> bit) & 1) == 0 else 0
__global__ void kFlagZeros(const uint32_t* __restrict__ in,
                           int n, int bit,
                           uint32_t* __restrict__ flagsZero) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;
    uint32_t x = in[i];
    uint32_t b = (x >> bit) & 1u;
    flagsZero[i] = (b ^ 1u);
}

// K2: per-block inclusive scan (Hillis–Steele), write exclusive result,
//     also write block sum.
__global__ void kBlockExclusiveScan(const uint32_t* __restrict__ in,
                                    uint32_t* __restrict__ exScan,
                                    uint32_t* __restrict__ blockSums,
                                    int n) {
    __shared__ uint32_t sh[BLOCK];

    int g0 = blockIdx.x * blockDim.x;
    int i  = g0 + threadIdx.x;

    uint32_t v = (i < n) ? in[i] : 0u;
    sh[threadIdx.x] = v;
    __syncthreads();

    for (int offset = 1; offset < BLOCK; offset <<= 1) {
        uint32_t t = 0u;
        if (threadIdx.x >= offset) t = sh[threadIdx.x - offset];
        __syncthreads();
        sh[threadIdx.x] += t;
        __syncthreads();
    }

    if (i < n) exScan[i] = sh[threadIdx.x] - v;

    if (threadIdx.x == BLOCK - 1) {
        int last = min(BLOCK, n - g0);
        uint32_t blkSum = (last > 0) ? sh[last - 1] : 0u;
        blockSums[blockIdx.x] = blkSum;
    }
}

// K3: add per-block offsets to make exScan global
__global__ void kAddBlockOffsets(uint32_t* __restrict__ exScan,
                                 const uint32_t* __restrict__ blockOffsets,
                                 int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;
    uint32_t off = blockOffsets[blockIdx.x];
    exScan[i] += off;
}

// K4: stable scatter by current bit
__global__ void kScatter(const uint32_t* __restrict__ in,
                         const uint32_t* __restrict__ exScanZero,
                         uint32_t totalZeros,
                         int n, int bit,
                         uint32_t* __restrict__ out) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;
    uint32_t x = in[i];
    uint32_t zBefore = exScanZero[i];
    uint32_t b = (x >> bit) & 1u;

    uint32_t pos = (b == 0u) ? zBefore
                             : (totalZeros + (uint32_t)i - zBefore);
    out[pos] = x;
}

extern "C" void radix_sort_1bit_host(unsigned int* data, int n) {
    if (n <= 0) return;

    uint32_t *bufA = nullptr, *bufB = nullptr;
    CK(hipMalloc(&bufA, n * sizeof(uint32_t)), "malloc bufA");
    CK(hipMalloc(&bufB, n * sizeof(uint32_t)), "malloc bufB");
    CK(hipMemcpy(bufA, data, n * sizeof(uint32_t), hipMemcpyDeviceToDevice), "copy input");

    uint32_t *d_flagsZero = nullptr;
    uint32_t *d_exScan    = nullptr;
    CK(hipMalloc(&d_flagsZero, n * sizeof(uint32_t)), "malloc flags");
    CK(hipMalloc(&d_exScan,    n * sizeof(uint32_t)), "malloc exScan");

    int numBlocks = (n + BLOCK - 1) / BLOCK;
    uint32_t *d_blockSums = nullptr, *d_blockOffsets = nullptr;
    CK(hipMalloc(&d_blockSums,    numBlocks * sizeof(uint32_t)), "malloc blockSums");
    CK(hipMalloc(&d_blockOffsets, numBlocks * sizeof(uint32_t)), "malloc blockOffsets");

    std::vector<uint32_t> h_block(numBlocks);

    for (int bit = 0; bit < 32; ++bit) {
        // flags
        kFlagZeros<<<numBlocks, BLOCK>>>(bufA, n, bit, d_flagsZero);
        CK(hipGetLastError(), "kFlagZeros");

        // per-block exscan
        kBlockExclusiveScan<<<numBlocks, BLOCK>>>(d_flagsZero, d_exScan, d_blockSums, n);
        CK(hipGetLastError(), "kBlockExclusiveScan");

        // host scan of block sums
        CK(hipMemcpy(h_block.data(), d_blockSums, numBlocks*sizeof(uint32_t), hipMemcpyDeviceToHost), "D2H blockSums");
        uint32_t totalZeros = 0, run = 0;
        for (int b = 0; b < numBlocks; ++b) {
            uint32_t s = h_block[b];
            h_block[b] = run;   // exclusive
            run += s;
        }
        totalZeros = run;
        CK(hipMemcpy(d_blockOffsets, h_block.data(), numBlocks*sizeof(uint32_t), hipMemcpyHostToDevice), "H2D blockOffsets");

        // add offsets
        kAddBlockOffsets<<<numBlocks, BLOCK>>>(d_exScan, d_blockOffsets, n);
        CK(hipGetLastError(), "kAddBlockOffsets");

        // scatter (stable)
        kScatter<<<numBlocks, BLOCK>>>(bufA, d_exScan, totalZeros, n, bit, bufB);
        CK(hipGetLastError(), "kScatter");

        CK(hipDeviceSynchronize(), "sync pass");
        std::swap(bufA, bufB);
    }

    CK(hipMemcpy(data, bufA, n*sizeof(uint32_t), hipMemcpyDeviceToDevice), "copy result");

    hipFree(bufA); hipFree(bufB);
    hipFree(d_flagsZero); hipFree(d_exScan);
    hipFree(d_blockSums); hipFree(d_blockOffsets);
}