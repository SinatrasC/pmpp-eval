#include <hip/hip_runtime.h>
#include <cstdio>
#include <vector>
#include <algorithm>
#include <cmath>
#include <cassert>

extern "C" void stencil25_stage1_boundary(const float* d_in, float* d_out,
                                          int dimx,int dimy,int dimz);

static inline size_t idx3(int i,int j,int k,int dx,int dy){
    return (size_t(k)*dy + j)*dx + i;
}

static void ck(hipError_t e, const char* m){
    if(e!=hipSuccess){ fprintf(stderr,"CUDA %s: %s\n", m, hipGetErrorString(e)); std::exit(2); }
}

static void fill_pattern(std::vector<float>& a){
    for(size_t i=0;i<a.size();++i){
        a[i] = sinf(0.013f*float(i)) + 0.001f*float((i*17)%101);
    }
}

static void cpu_stage1_oracle(const std::vector<float>& in, std::vector<float>& out,
                              int dimx,int dimy,int dimz)
{
    const int R=4;
    const int tot_z = dimz + 8;
    const int zBeg = 4;
    const int zEnd = 4 + dimz - 1;

    const float w0=0.5f, w1=0.10f, w2=0.05f, w3=0.025f, w4=0.0125f;
    const float w[5]={w0,w1,w2,w3,w4};

    out = in; // initialize to in; we will overwrite Stage-1 planes; others remain as-is

    auto compute = [&](int i,int j,int k)->float{
        float acc = w[0]*in[idx3(i,j,k,dimx,dimy)];
        for(int d=1; d<=4; ++d){
            acc += w[d]*( in[idx3(i-d,j,k,dimx,dimy)]+in[idx3(i+d,j,k,dimx,dimy)]
                        + in[idx3(i,j-d,k,dimx,dimy)]+in[idx3(i,j+d,k,dimx,dimy)]
                        + in[idx3(i,j,k-d,dimx,dimy)]+in[idx3(i,j,k+d,dimx,dimy)] );
        }
        return acc;
    };

    // left boundary planes
    for(int k=zBeg; k<=zBeg+3; ++k){
        for(int j=0;j<dimy;++j){
            for(int i=0;i<dimx;++i){
                size_t p=idx3(i,j,k,dimx,dimy);
                if(i>=R && i<dimx-R && j>=R && j<dimy-R) out[p]=compute(i,j,k);
                else out[p]=in[p];
            }
        }
    }
    // right boundary planes
    for(int k=zEnd-3; k<=zEnd; ++k){
        for(int j=0;j<dimy;++j){
            for(int i=0;i<dimx;++i){
                size_t p=idx3(i,j,k,dimx,dimy);
                if(i>=R && i<dimx-R && j>=R && j<dimy-R) out[p]=compute(i,j,k);
                else out[p]=in[p];
            }
        }
    }
}

static bool almost_equal(const std::vector<float>& a, const std::vector<float>& b){
    if(a.size()!=b.size()) return false;
    for(size_t i=0;i<a.size();++i){
        float A=a[i], B=b[i];
        float diff=fabsf(A-B);
        float tol = 1e-5f + 1e-5f*std::max(fabsf(A),fabsf(B));
        if(diff>tol) return false;
    }
    return true;
}

int main(){
    printf("ch20-stencil-25pt-slab-stage1-boundary tests\n");
    struct C{int x,y,z;};
    const C cases[] = {{16,16,8}, {32,24,10}, {48,48,40}};

    int total=0, pass=0;

    for(const auto& cs: cases){
        ++total;
        int dimx=cs.x, dimy=cs.y, dimz=cs.z;
        int totz=dimz+8;
        size_t N = size_t(dimx)*dimy*totz;

        // canary guard
        const size_t GUARD=4096;
        const float SENT=1337.0f;

        std::vector<float> hin(N), href(N), hout(N,0.0f);
        fill_pattern(hin);
        cpu_stage1_oracle(hin, href, dimx,dimy,dimz);

        std::vector<float> h_in_guard(N+2*GUARD,SENT);
        std::copy(hin.begin(),hin.end(),h_in_guard.begin()+GUARD);

        std::vector<float> h_out_guard(N+2*GUARD,SENT);

        float *d_in_all=nullptr,*d_out_all=nullptr;
        ck(hipMalloc(&d_in_all, (N+2*GUARD)*sizeof(float)),"malloc in");
        ck(hipMalloc(&d_out_all,(N+2*GUARD)*sizeof(float)),"malloc out");
        ck(hipMemcpy(d_in_all,h_in_guard.data(),(N+2*GUARD)*sizeof(float),hipMemcpyHostToDevice),"H2D in");
        ck(hipMemcpy(d_out_all,h_out_guard.data(),(N+2*GUARD)*sizeof(float),hipMemcpyHostToDevice),"H2D out");

        float* d_in = d_in_all + GUARD;
        float* d_out= d_out_all+ GUARD;

        // start with d_out = d_in (so non-Stage1 planes pass through unchanged)
        ck(hipMemcpy(d_out, d_in, N*sizeof(float), hipMemcpyDeviceToDevice), "seed out");

        stencil25_stage1_boundary(d_in, d_out, dimx,dimy,dimz);

        ck(hipMemcpy(h_out_guard.data(), d_out_all,(N+2*GUARD)*sizeof(float), hipMemcpyDeviceToHost),"D2H out");
        std::copy(h_out_guard.begin()+GUARD, h_out_guard.begin()+GUARD+N, hout.begin());

        auto guard_ok=[&](const std::vector<float>& g){
            for(size_t t=0;t<GUARD;t++){
                if(g[t]!=SENT || g[g.size()-1-t]!=SENT) return false;
            } return true;
        };

        bool ok = almost_equal(hout, href) && guard_ok(h_out_guard);
        printf("Case %3dx%3dx%3d -> %s\n", dimx,dimy,dimz, ok?"OK":"FAIL");
        if(ok) ++pass;

        hipFree(d_in_all); hipFree(d_out_all);
    }

    printf("Summary: %d/%d passed\n", pass,total);
    return (pass==total)?0:1;
}