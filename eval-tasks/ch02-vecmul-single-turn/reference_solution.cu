#include <hip/hip_runtime.h>

__global__ void vecMulKernel(const float* A, const float* B, float* C, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) C[i] = A[i] * B[i];
}