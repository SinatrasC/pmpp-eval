// ch15-bfs-push-single / test_bfs_push.cu
#include <hip/hip_runtime.h>
#include <vector>
#include <queue>
#include <algorithm>
#include <cstdio>
#include <cstdlib>
#include <limits>
#include <cassert>
#include <random>

extern "C" void bfs_push_gpu(const int* d_row_ptr,
                             const int* d_col_idx,
                             int V, int E,
                             int src,
                             int* d_level);

static void ck(hipError_t e, const char* m){
    if(e!=hipSuccess){ fprintf(stderr,"CUDA %s: %s\n", m, hipGetErrorString(e)); std::exit(2); }
}

struct CSR {
    int V=0; int E=0;
    std::vector<int> row_ptr; // V+1
    std::vector<int> col_idx; // E
};

static CSR make_chain(int V){
    CSR g; g.V=V; g.row_ptr.resize(V+1,0);
    std::vector<std::vector<int>> adj(V);
    for(int i=0;i<V-1;i++){ adj[i].push_back(i+1); adj[i+1].push_back(i); }
    int E=0; for(int i=0;i<V;i++){ E+= (int)adj[i].size(); }
    g.E=E; g.col_idx.resize(E);
    int off=0; for(int i=0;i<V;i++){ g.row_ptr[i]=off; for(int v:adj[i]) g.col_idx[off++]=v; }
    g.row_ptr[V]=off; return g;
}

static CSR make_star(int V){
    CSR g; g.V=V; g.row_ptr.resize(V+1,0);
    std::vector<std::vector<int>> adj(V);
    if(V>0){ for(int i=1;i<V;i++){ adj[0].push_back(i); adj[i].push_back(0);} }
    int E=0; for(int i=0;i<V;i++) E+=(int)adj[i].size();
    g.E=E; g.col_idx.resize(E);
    int off=0; for(int i=0;i<V;i++){ g.row_ptr[i]=off; for(int v:adj[i]) g.col_idx[off++]=v; }
    g.row_ptr[V]=off; return g;
}

static CSR make_two_components(int a, int b){
    CSR g; g.V=a+b; g.row_ptr.resize(g.V+1,0);
    std::vector<std::vector<int>> adj(g.V);
    for(int i=0;i<a-1;i++){ adj[i].push_back(i+1); adj[i+1].push_back(i); }
    for(int i=0;i<b-1;i++){ int u=a+i, v=a+i+1; adj[u].push_back(v); adj[v].push_back(u); }
    int E=0; for(int i=0;i<g.V;i++) E+=(int)adj[i].size();
    g.E=E; g.col_idx.resize(E);
    int off=0; for(int i=0;i<g.V;i++){ g.row_ptr[i]=off; for(int v:adj[i]) g.col_idx[off++]=v; }
    g.row_ptr[g.V]=off; return g;
}

static CSR make_grid2d(int W,int H){
    int V=W*H; auto id=[&](int x,int y){return y*W+x;};
    CSR g; g.V=V; g.row_ptr.resize(V+1,0);
    std::vector<std::vector<int>> adj(V);
    for(int y=0;y<H;y++)for(int x=0;x<W;x++){
        int u=id(x,y);
        if(x+1<W){ adj[u].push_back(id(x+1,y)); adj[id(x+1,y)].push_back(u); }
        if(y+1<H){ adj[u].push_back(id(x,y+1)); adj[id(x,y+1)].push_back(u); }
    }
    int E=0; for(int i=0;i<V;i++) E+=(int)adj[i].size();
    g.E=E; g.col_idx.resize(E);
    int off=0; for(int i=0;i<V;i++){ g.row_ptr[i]=off; for(int v:adj[i]) g.col_idx[off++]=v; }
    g.row_ptr[V]=off; return g;
}

static CSR make_erdos(int V, float p, unsigned seed=123){
    CSR g; g.V=V; g.row_ptr.resize(V+1,0);
    std::vector<std::vector<int>> adj(V);
    std::mt19937 rng(seed); std::uniform_real_distribution<float> U(0,1);
    for(int i=0;i<V;i++) for(int j=i+1;j<V;j++) if(U(rng)<p){ adj[i].push_back(j); adj[j].push_back(i); }
    int E=0; for(int i=0;i<V;i++) E+=(int)adj[i].size();
    g.E=E; g.col_idx.resize(E);
    int off=0; for(int i=0;i<V;i++){ g.row_ptr[i]=off; for(int v:adj[i]) g.col_idx[off++]=v; }
    g.row_ptr[V]=off; return g;
}

static std::vector<int> cpu_bfs_levels(const CSR& g, int src){
    const int INF = 0x3f3f3f3f;
    std::vector<int> lvl(g.V, INF);
    if(g.V==0) return lvl;
    std::queue<int>q; lvl[src]=0; q.push(src);
    while(!q.empty()){
        int u=q.front(); q.pop();
        for(int e=g.row_ptr[u]; e<g.row_ptr[u+1]; ++e){
            int v=g.col_idx[e];
            if(lvl[v]==INF){ lvl[v]=lvl[u]+1; q.push(v); }
        }
    }
    return lvl;
}

int main(){
    printf("ch15-bfs-push-single tests\n");

    std::vector<CSR> gs;
    gs.push_back(make_chain(1));
    gs.push_back(make_chain(10));
    gs.push_back(make_star(101));
    gs.push_back(make_two_components(10,15));
    gs.push_back(make_grid2d(8,8));   // 64
    gs.push_back(make_grid2d(16,16)); // 256
    gs.push_back(make_erdos(200, 0.03f, 777));

    const int SENT = 0xDEADBEEF;
    int total=0, passed=0;

    for(size_t gi=0; gi<gs.size(); ++gi){
        const CSR& g = gs[gi];
        int V=g.V, E=g.E, src=0;
        ++total;

        // CPU oracle
        auto ref = cpu_bfs_levels(g, src);

        // Guarded device buffers
        size_t GU = 1024;
        int *d_row_all=nullptr, *d_col_all=nullptr, *d_lvl_all=nullptr;
        ck(hipMalloc(&d_row_all, (V+1+2*GU)*sizeof(int)), "malloc row");
        ck(hipMalloc(&d_col_all, (E  +2*GU)*sizeof(int)), "malloc col");
        ck(hipMalloc(&d_lvl_all, (V  +2*GU)*sizeof(int)), "malloc lvl");

        std::vector<int> h_row_guard(V+1+2*GU, SENT);
        std::vector<int> h_col_guard(E  +2*GU, SENT);
        std::vector<int> h_lvl_guard(V  +2*GU, SENT);

        if(V>0){ std::copy(g.row_ptr.begin(), g.row_ptr.end(), h_row_guard.begin()+GU); }
        if(E>0){ std::copy(g.col_idx.begin(), g.col_idx.end(), h_col_guard.begin()+GU); }

        ck(hipMemcpy(d_row_all, h_row_guard.data(), (V+1+2*GU)*sizeof(int), hipMemcpyHostToDevice), "H2D row");
        ck(hipMemcpy(d_col_all, h_col_guard.data(), (E  +2*GU)*sizeof(int), hipMemcpyHostToDevice), "H2D col");
        ck(hipMemcpy(d_lvl_all, h_lvl_guard.data(), (V  +2*GU)*sizeof(int), hipMemcpyHostToDevice), "H2D lvl");

        int* d_row = d_row_all + GU;
        int* d_col = d_col_all + GU;
        int* d_lvl = d_lvl_all + GU;

        // RUN
        bfs_push_gpu(d_row, d_col, V, E, src, d_lvl);
        ck(hipDeviceSynchronize(), "sync");

        // Download
        ck(hipMemcpy(h_row_guard.data(), d_row_all, (V+1+2*GU)*sizeof(int), hipMemcpyDeviceToHost), "D2H row");
        ck(hipMemcpy(h_col_guard.data(), d_col_all, (E  +2*GU)*sizeof(int), hipMemcpyDeviceToHost), "D2H col");
        ck(hipMemcpy(h_lvl_guard.data(), d_lvl_all, (V  +2*GU)*sizeof(int), hipMemcpyDeviceToHost), "D2H lvl");

        // Extract level
        std::vector<int> got(V, SENT);
        if(V>0) std::copy(h_lvl_guard.begin()+GU, h_lvl_guard.begin()+GU+V, got.begin());

        // Checks
        auto guards_ok = [&](const std::vector<int>& gbuf){
            for(size_t t=0;t<GU;t++){ if(gbuf[t]!=SENT || gbuf[gbuf.size()-1-t]!=SENT) return false; }
            return true;
        };

        bool ok = true;
        // CSR input immutability
        if(V>0){
            std::vector<int> row_in(V+1);
            std::copy(h_row_guard.begin()+GU, h_row_guard.begin()+GU+V+1, row_in.begin());
            ok = ok && (row_in == g.row_ptr);
        }
        if(E>0){
            std::vector<int> col_in(E);
            std::copy(h_col_guard.begin()+GU, h_col_guard.begin()+GU+E, col_in.begin());
            ok = ok && (col_in == g.col_idx);
        }
        ok = ok && guards_ok(h_row_guard) && guards_ok(h_col_guard) && guards_ok(h_lvl_guard);
        ok = ok && (got == ref);

        printf("Graph %zu (V=%d,E=%d) -> %s\n", gi, V, E, ok?"OK":"FAIL");
        if(ok) ++passed;

        hipFree(d_row_all); hipFree(d_col_all); hipFree(d_lvl_all);
    }

    printf("Summary: %d/%d passed\n", passed, total);
    return (passed==total)?0:1;
}