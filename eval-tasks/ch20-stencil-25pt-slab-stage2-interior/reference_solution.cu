#include <hip/hip_runtime.h>
#include <cstdio>
#include <vector>
#include <algorithm>
#include <cmath>
#include <cassert>

static inline __host__ __device__
size_t idx3(int i,int j,int k,int dx,int dy){ return (size_t(k)*dy + j)*dx + i; }

__global__ void stencil25_stage2_kernel_ref(const float* __restrict__ in,
                                            float* __restrict__ out,
                                            int dimx,int dimy,int dimz)
{
    const int R=4;
    const int zOwnedBeg = 4;
    const int zOwnedEnd = 4 + dimz - 1;

    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;
    int k = blockIdx.z*blockDim.z + threadIdx.z; // local z including halos

    if(i>=dimx || j>=dimy || k>=dimz+8) return;

    // Stage-2 interior planes only
    if( !(k >= (zOwnedBeg+4) && k <= (zOwnedEnd-4)) ) return;

    const bool interiorXY = (i>=R && i<dimx-R && j>=R && j<dimy-R);
    const size_t p = idx3(i,j,k,dimx,dimy);

    if(!interiorXY){
        // copy-through on x/y edges inside the interior z band
        out[p] = in[p];
        return;
    }

    const float w0=0.5f, w1=0.10f, w2=0.05f, w3=0.025f, w4=0.0125f;
    const float w[5]={w0,w1,w2,w3,w4};

    float acc = w[0]*in[p];
    #pragma unroll
    for(int d=1; d<=4; ++d){
        acc += w[d] * ( in[idx3(i-d,j,k,dimx,dimy)] + in[idx3(i+d,j,k,dimx,dimy)]
                      + in[idx3(i,j-d,k,dimx,dimy)] + in[idx3(i,j+d,k,dimx,dimy)]
                      + in[idx3(i,j,k-d,dimx,dimy)] + in[idx3(i,j,k+d,dimx,dimy)] );
    }
    out[p] = acc;
}

extern "C" void stencil25_stage2_interior(const float* d_in, float* d_out,
                                          int dimx,int dimy,int dimz)
{
    dim3 block(8,8,8);
    dim3 grid( (dimx+block.x-1)/block.x,
               (dimy+block.y-1)/block.y,
               ((dimz+8)+block.z-1)/block.z );
    stencil25_stage2_kernel_ref<<<grid,block>>>(d_in, d_out, dimx,dimy,dimz);
    hipDeviceSynchronize();
}