#include <hip/hip_runtime.h>
#include <cstdio>
#include <vector>
#include <algorithm>
#include <cmath>
#include <cassert>

extern "C" void stencil25_stage2_interior(const float* d_in, float* d_out,
                                          int dimx,int dimy,int dimz);

static inline size_t idx3(int i,int j,int k,int dx,int dy){ return (size_t(k)*dy + j)*dx + i; }

static void ck(hipError_t e,const char* m){ if(e!=hipSuccess){fprintf(stderr,"CUDA %s: %s\n",m,hipGetErrorString(e)); std::exit(2);} }

static void fill_pattern(std::vector<float>& a){
    for(size_t t=0;t<a.size();++t){
        a[t] = cosf(0.007f*float(t)) + 0.002f*float((t*31)%113);
    }
}

static void cpu_stage2_oracle(const std::vector<float>& in, std::vector<float>& out,
                              int dimx,int dimy,int dimz)
{
    const int R=4;
    const int zOwnedBeg = 4;
    const int zOwnedEnd = 4 + dimz - 1;
    const int totz = dimz + 8;

    out = in; // untouched by default

    auto compute=[&](int i,int j,int k)->float{
        const float w0=0.5f, w1=0.10f, w2=0.05f, w3=0.025f, w4=0.0125f;
        const float w[5]={w0,w1,w2,w3,w4};
        float acc = w[0]*in[idx3(i,j,k,dimx,dimy)];
        for(int d=1; d<=4; ++d){
            acc += w[d]*( in[idx3(i-d,j,k,dimx,dimy)] + in[idx3(i+d,j,k,dimx,dimy)]
                        + in[idx3(i,j-d,k,dimx,dimy)] + in[idx3(i,j+d,k,dimx,dimy)]
                        + in[idx3(i,j,k-d,dimx,dimy)] + in[idx3(i,j,k+d,dimx,dimy)] );
        }
        return acc;
    };

    for(int k=zOwnedBeg+4; k<=zOwnedEnd-4; ++k){
        for(int j=0;j<dimy;++j){
            for(int i=0;i<dimx;++i){
                size_t p=idx3(i,j,k,dimx,dimy);
                if(i>=R && i<dimx-R && j>=R && j<dimy-R) out[p]=compute(i,j,k);
                else out[p]=in[p];
            }
        }
    }
}

static bool almost_equal(const std::vector<float>& a, const std::vector<float>& b){
    if(a.size()!=b.size()) return false;
    for(size_t i=0;i<a.size();++i){
        float A=a[i], B=b[i];
        float diff=fabsf(A-B);
        float tol=1e-5f + 1e-5f*std::max(fabsf(A),fabsf(B));
        if(diff>tol) return false;
    }
    return true;
}

int main(){
    printf("ch20-stencil-25pt-slab-stage2-interior tests\n");
    struct C{int x,y,z;};
    const C cases[]={{16,16,8},{32,24,10},{48,48,40}};

    int total=0, pass=0;

    for(const auto& cs: cases){
        ++total;
        int dimx=cs.x, dimy=cs.y, dimz=cs.z;
        int totz=dimz+8;
        size_t N=size_t(dimx)*dimy*totz;

        const size_t GUARD=4096;
        const float SENT=1337.0f;

        std::vector<float> hin(N), href(N), hout(N);
        fill_pattern(hin);
        cpu_stage2_oracle(hin, href, dimx,dimy,dimz);

        std::vector<float> h_in_guard(N+2*GUARD,SENT), h_out_guard(N+2*GUARD,SENT);
        std::copy(hin.begin(),hin.end(),h_in_guard.begin()+GUARD);

        float *d_in_all=nullptr,*d_out_all=nullptr;
        ck(hipMalloc(&d_in_all, (N+2*GUARD)*sizeof(float)),"malloc in");
        ck(hipMalloc(&d_out_all,(N+2*GUARD)*sizeof(float)),"malloc out");
        ck(hipMemcpy(d_in_all, h_in_guard.data(), (N+2*GUARD)*sizeof(float), hipMemcpyHostToDevice),"H2D in");
        ck(hipMemcpy(d_out_all,h_out_guard.data(),(N+2*GUARD)*sizeof(float), hipMemcpyHostToDevice),"H2D out");

        float* d_in = d_in_all + GUARD;
        float* d_out= d_out_all+ GUARD;

        // Seed out = in (so non-updated planes remain pass-through)
        ck(hipMemcpy(d_out, d_in, N*sizeof(float), hipMemcpyDeviceToDevice), "seed out");

        stencil25_stage2_interior(d_in, d_out, dimx,dimy,dimz);

        ck(hipMemcpy(h_out_guard.data(), d_out_all,(N+2*GUARD)*sizeof(float), hipMemcpyDeviceToHost),"D2H out");
        std::copy(h_out_guard.begin()+GUARD, h_out_guard.begin()+GUARD+N, hout.begin());

        auto guard_ok=[&](const std::vector<float>& g){
            for(size_t t=0;t<GUARD;t++){
                if(g[t]!=SENT || g[g.size()-1-t]!=SENT) return false;
            } return true;
        };

        bool ok = almost_equal(hout, href) && guard_ok(h_out_guard);
        printf("Case %3dx%3dx%3d -> %s\n", dimx,dimy,dimz, ok?"OK":"FAIL");
        if(ok) ++pass;

        hipFree(d_in_all); hipFree(d_out_all);
    }

    printf("Summary: %d/%d passed\n", pass,total);
    return (pass==total)?0:1;
}